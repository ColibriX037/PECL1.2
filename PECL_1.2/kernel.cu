#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <windows.h>


using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

__device__ void add_up(int *matriz, int x, int y, int altura, int anchura)
{
	if (x != 0 && y < anchura)
	{
		//printf("Soy el hilo alturaa %d id %d valor %d\n", x, y, matriz[x*anchura + y]);
		if (matriz[x*anchura + y] != 0)
		{
			//printf("Soy el hilo alturaa %d id %d valor %d distinto de cero\n", x, y, matriz[x*anchura + y]);
			if (matriz[x*anchura + y] == matriz[(x - 1)*anchura + y])
			{
				//printf("Soy el hilo alturaa %d id %d valor %d y mi anterior hilo alturaa %d id %d valor %d es igual que yo\n", x, y, matriz[x*anchura + y], x, y - 1, matriz[x*anchura + (y - 1)]);
				int iguales = 0;
				iguales++;
				for (int i = 1; i <= x; i++)
				{
					if (matriz[x*anchura + y] == matriz[(x - i)*anchura + y])
					{
						iguales++;
					}
					else {
						break;
					}
				}
				if (iguales % 2 == 0)
				{
					matriz[(x - 1)*anchura + y] = matriz[(x - 1)*anchura + y] * 2;
					matriz[x*anchura + y] = 0;
				}
			}
			else if (matriz[(x - 1)*anchura + y] == 0)
			{
				matriz[(x - 1)*anchura + y] = matriz[x*anchura + y];
				matriz[x*anchura + y] = 0;
			}
		}
	}
}

__device__ void stack_up(int *matriz, int anchura, int altura, int x, int y) {
	//printf("soy el hilo x%d y%d y empiezo a ejecutar\n", x, y);
	for (int i = altura - 1; i > 0; i--)
	{
		if ((x != 0) && (matriz[x*anchura + y] != 0) && matriz[x*anchura + (y - anchura)] == 0)
		{
			//printf("soy el hilo x%d y%d y el de mi izquierda es un 0\n", x, y);
			matriz[x*anchura + (y - anchura)] = matriz[x*anchura + y];
			matriz[x*anchura + y] = 0;
		}
		__syncthreads();
	}
}

__global__ void mov_upK(int *matriz, int anchura, int altura) {
	int x = threadIdx.x;
	int y = threadIdx.y;

	stack_up(matriz, anchura, altura, x, y);
	add_up(matriz, x, y, altura, anchura);
	__syncthreads();
	stack_up(matriz, anchura, altura, x, y);
}


hipError_t move_up(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 dimgrid(1, 1);
	dim3 dimblock(alto, ancho, 1);

	mov_upK << < dimgrid, dimblock >> > (dev_m, ancho, alto);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__device__ void add_down(int *matriz, int x, int y, int altura, int anchura)
{
	if (x != altura - 1 && y < anchura)
	{
		//printf("Soy el hilo alturaa %d id %d valor %d\n", x, y, matriz[x*anchura + y]);
		if (matriz[x*anchura + y] != 0)
		{
			//printf("Soy el hilo alturaa %d id %d valor %d distinto de cero\n", x, y, matriz[x*anchura + y]);
			if (matriz[x*anchura + y] == matriz[(x + 1)*anchura + y])
			{
				//printf("Soy el hilo alturaa %d id %d valor %d y mi anterior hilo alturaa %d id %d valor %d es igual que yo\n", x, y, matriz[x*anchura + y], x, y - 1, matriz[x*anchura + (y - 1)]);
				int iguales = 0;
				iguales++;
				for (int i = 1; x + i <= altura; i++)
				{
					if (matriz[x*anchura + y] == matriz[(x + i)*anchura + y])
					{
						iguales++;
					}
					else {
						break;
					}
				}
				if (iguales % 2 == 0)
				{
					matriz[(x + 1)*anchura + y] = matriz[(x + 1)*anchura + y] * 2;
					matriz[x*anchura + y] = 0;
				}
			}
			else if (matriz[(x + 1)*anchura + y] == 0)
			{
				matriz[(x + 1)*anchura + y] = matriz[x*anchura + y];
				matriz[x*anchura + y] = 0;
			}
		}
	}
}

__device__ void stack_down(int *matriz, int anchura, int altura, int x, int y) {
	//printf("soy el hilo x%d y%d y empiezo a ejecutar\n", x, y);
	for (int i = altura - 1; i > 0; i--)
	{
		if ((x != altura - 1) && (matriz[x*anchura + y] != 0) && matriz[(x + 1)*anchura + y] == 0)
		{
			//printf("soy el hilo x%d y%d y el de mi izquierda es un 0\n", x, y);
			matriz[(x + 1)*anchura + y] = matriz[x*anchura + y];
			matriz[x*anchura + y] = 0;
		}
		__syncthreads();
	}
}

__global__ void mov_downK(int *matriz, int anchura, int altura) {
	int x = threadIdx.x;
	int y = threadIdx.y;

	stack_down(matriz, anchura, altura, x, y);
	add_down(matriz, x, y, altura, anchura);
	__syncthreads();
	stack_down(matriz, anchura, altura, x, y);
}

hipError_t move_down(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 dimgrid(1, 1);
	dim3 dimblock(alto, ancho, 1);

	mov_downK << < dimgrid, dimblock >> > (dev_m, ancho, alto);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__device__ void add_left(int *matriz, int x, int y, int altura, int anchura)
{
	if (y != 0 && y < anchura)
	{
		//printf("Soy el hilo alturaa %d id %d valor %d\n", x, y, matriz[x*anchura + y]);
		if (matriz[x*anchura + y] != 0)
		{
			//printf("Soy el hilo alturaa %d id %d valor %d distinto de cero\n", x, y, matriz[x*anchura + y]);
			if (matriz[x*anchura + y] == matriz[x*anchura + (y - 1)])
			{
				//printf("Soy el hilo alturaa %d id %d valor %d y mi anterior hilo alturaa %d id %d valor %d es igual que yo\n", x, y, matriz[x*anchura + y], x, y - 1, matriz[x*anchura + (y - 1)]);
				int iguales = 0;
				iguales++;
				for (int i = 1; i <= y; i++)
				{
					if (matriz[x*anchura + y] == matriz[x*anchura + (y - i)])
					{
						iguales++;
					}
					else {
						break;
					}
				}
				if (iguales % 2 == 0)
				{
					matriz[x*anchura + (y - 1)] = matriz[x*anchura + (y - 1)] * 2;
					matriz[x*anchura + y] = 0;
				}
			}
			else if (matriz[x*anchura + (y - 1)] == 0)
			{
				matriz[x*anchura + (y - 1)] = matriz[x*anchura + y];
				matriz[x*anchura + y] = 0;
			}
		}
	}
}

__device__ void stack_left(int *matriz, int anchura, int altura, int x, int y) {

	//printf("soy el hilo x%d y%d y empiezo a ejecutar\n", x, y);
	for (int i = anchura - 1; i > 0; i--)
	{
		if ((y != 0) && (matriz[x*anchura + y] != 0) && matriz[x*anchura + (y - 1)] == 0)
		{
			//printf("soy el hilo x%d y%d y el de mi izquierda es un 0\n", x, y);
			matriz[x*anchura + (y - 1)] = matriz[x*anchura + y];
			matriz[x*anchura + y] = 0;
		}
		__syncthreads();
	}
}

__global__ void mov_leftK(int *matriz, int anchura, int altura) {
	int x = threadIdx.x;
	int y = threadIdx.y;

	stack_left(matriz, anchura, altura, x, y);
	add_left(matriz, x, y, altura, anchura);
	__syncthreads();
	stack_left(matriz, anchura, altura, x, y);
}

hipError_t move_left(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 dimgrid(1, 1);
	dim3 dimblock(alto, ancho, 1);

	mov_leftK << < dimgrid, dimblock >> > (dev_m, ancho, alto);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}

__device__ void add_right(int *matriz, int x, int y, int altura, int anchura)
{
	if (y != anchura - 1 && y < anchura)
	{
		//printf("Soy el hilo alturaa %d id %d valor %d\n", x, y, matriz[x*anchura + y]);
		if (matriz[x*anchura + y] != 0)
		{
			//printf("Soy el hilo alturaa %d id %d valor %d distinto de cero\n", x, y, matriz[x*anchura + y]);
			if (matriz[x*anchura + y] == matriz[x*anchura + (y + 1)])
			{
				//printf("Soy el hilo alturaa %d id %d valor %d y mi anterior hilo alturaa %d id %d valor %d es igual que yo\n", x, y, matriz[x*anchura + y], x, y - 1, matriz[x*anchura + (y - 1)]);
				int iguales = 0;
				iguales++;
				for (int i = 1; y + i < anchura; i++)
				{
					if (matriz[x*anchura + y] == matriz[x*anchura + (y + i)])
					{
						iguales++;
					}
					else {
						break;
					}
				}
				if (iguales % 2 == 0)
				{
					matriz[x*anchura + (y + 1)] = matriz[x*anchura + (y + 1)] * 2;
					matriz[x*anchura + y] = 0;
				}
			}
			else if (matriz[x*anchura + (y + 1)] == 0)
			{
				matriz[x*anchura + (y + 1)] = matriz[x*anchura + y];
				matriz[x*anchura + y] = 0;
			}
		}
	}
}
__device__ void stack_right(int *matriz, int anchura, int altura, int x, int y) {

	//printf("soy el hilo x%d y%d y empiezo a ejecutar\n", x, y);
	for (int i = anchura - 1; i > 0; i--)
	{
		if ((y != anchura - 1) && (matriz[x*anchura + y] != 0) && matriz[x*anchura + (y + 1)] == 0)
		{
			//printf("soy el hilo x%d y%d y el de mi izquierda es un 0\n", x, y);
			matriz[x*anchura + (y + 1)] = matriz[x*anchura + y];
			matriz[x*anchura + y] = 0;
		}
		__syncthreads();
	}
}

__global__ void mov_rightK(int *matriz, int anchura, int altura) {
	int x = threadIdx.x;
	int y = threadIdx.y;

	stack_right(matriz, anchura, altura, x, y);
	add_right(matriz, x, y, altura, anchura);
	__syncthreads();
	stack_right(matriz, anchura, altura, x, y);
}



hipError_t move_right(int *matriz, int ancho, int alto) {
	hipError_t cudaStatus;

	int *dev_m;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en setdevice");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_m, ancho*alto * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en Malloc");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_m, matriz, ancho*alto * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 dimgrid(1, 1);
	dim3 dimblock(alto, ancho, 1);

	mov_rightK << < dimgrid, dimblock >> > (dev_m, ancho, alto);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en mov_upK");
		goto Error;
	}

	cudaStatus = hipMemcpy(matriz, dev_m, ancho*alto * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Error en memcpy to host de mov_upK");
		goto Error;
	}

Error:
	hipFree(dev_m);

	return cudaStatus;
}


int main()
{
	hipError_t cudaStatus;
	srand(time(NULL));

	int ancho;
	int alto;
	int numSemillas = 0;
	int vidas = 5;
	char modo;
	char cargado;
	char ia;
	int *datos;
	int *matriz;
	int *especificaciones;

	especificaciones = MostrarEspecificaciones();

	printf("Desea activar la IA? (y/n)");
	cin >> ia;

	printf("Desea comprobar si hay partidas guardadas?(y/n): ");
	cin >> cargado;
	if (cargado == 'y')
	{
		datos = cargar();

		vidas = datos[0];
		alto = datos[1];
		ancho = datos[2];

		int dificultad = datos[3];

		if (dificultad == 0)
		{
			modo = 'B';
			numSemillas = 15;
		}
		else
		{
			modo = 'A';
			numSemillas = 8;
		}

		matriz = (int*)malloc(ancho*alto * sizeof(int));

		for (int i = 0; i < alto*ancho; i++)
		{
			matriz[i] = datos[4 + i];
		}
	}
	else
	{
		printf("Indique el ancho de la matriz: ");
		cin >> ancho;
		printf("Indique el alto de la matriz: ");
		cin >> alto;

		if (alto*ancho > especificaciones[0])
		{
			printf("La matriz seleccionada es demasiado grande para tu tarjeta grafica. Lo siento.");
			return 0;
		}


		printf("Indique la dificultad del juego (B->Bajo / A->Alto): ");
		cin >> modo;
		switch (modo)
		{
		case 'B':
			numSemillas = 15;
			break;
		case 'A':
			numSemillas = 8;
			break;
		default:
			break;
		}



		matriz = (int*)malloc(ancho*alto * sizeof(int));
		for (int i = 0; i < ancho*alto; i++) {
			matriz[i] = 0;
		}
	}

	if (ia == 'n')
	{
		while ((!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
		{
			//system("CLS");



			if (!(!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
			{
				for (int i = 0; i < ancho*alto; i++) {
					matriz[i] = 0;
				}
				vidas--;
			}




			gestionSemillas(matriz, ancho, numSemillas, alto, modo);

			printf("checkMove: %d\n", checkMove(matriz, ancho, alto));
			printf("checkFull: %d\n", checkFull(matriz, ancho*alto));

			char movimiento = 'p';
			printf("Vidas restantes: %d\n", vidas);
			printf("Tablero:\n");
			showMatriz(matriz, ancho, alto);
			printf("Hacia donde quieres mover?(w/a/s/d) Para guardar teclee g: ");
			cin >> movimiento;
			switch (movimiento)
			{
			case 'w':
				cudaStatus = move_up(matriz, ancho, alto);
				break;
			case 'a':
				cudaStatus = move_left(matriz, ancho, alto);
				break;
			case 's':
				cudaStatus = move_down(matriz, ancho, alto);
				break;
			case 'd':
				cudaStatus = move_right(matriz, ancho, alto);
				break;
			case 'g':
				guardar(vidas, matriz, alto, ancho, modo);
				printf("Partida guardada, hasta pronto!");
				return 0;
			default:
				break;
			}


			if (!(!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
			{
				for (int i = 0; i < ancho*alto; i++) {
					matriz[i] = 0;
				}
				vidas--;
			}

		}



		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
	}
	else {

		while ((!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
		{
			if (!(!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
			{
				for (int i = 0; i < ancho*alto; i++) {
					matriz[i] = 0;
				}
				vidas--;
			}


			system("CLS");

			gestionSemillas(matriz, ancho, numSemillas, alto, modo);

			printf("checkMove: %d\n", checkMove(matriz, ancho, alto));
			printf("checkFull: %d\n", checkFull(matriz, ancho*alto));

			char movimiento = 'p';
			printf("Vidas restantes: %d\n", vidas);
			printf("Tablero:\n");
			showMatriz(matriz, ancho, alto);

			int r = rand() % 4;

			switch (r)
			{
			case 0:
				printf("Moviendo hacia arriba\n");
				cudaStatus = move_up(matriz, ancho, alto);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceReset failed!");
					return 1;
				}
				break;
			case 1:
				printf("Moviendo hacia izquierda\n");
				cudaStatus = move_left(matriz, ancho, alto);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceReset failed!");
					return 1;
				}
				break;
			case 2:
				printf("Moviendo hacia abajo\n");
				cudaStatus = move_down(matriz, ancho, alto);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceReset failed!");
					return 1;
				}
				break;
			case 3:
				printf("Moviendo hacia derecha\n");
				cudaStatus = move_right(matriz, ancho, alto);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipDeviceReset failed!");
					return 1;
				}
				break;
			default:
				break;
			}
			//Sleep(100);

			if (!(!checkFull(matriz, ancho*alto) || checkMove(matriz, ancho, alto)) && vidas > 0)
			{
				for (int i = 0; i < ancho*alto; i++) {
					matriz[i] = 0;
				}
				vidas--;
			}

		}



		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
	}


	return 0;
}

// Metodo que SOLO muestra matrices cuadradas
void showMatriz(int *matriz, int anchura, int altura)
{
	for (int i = 0; i < altura; i++)
	{
		for (int j = 0; j < anchura; j++)
		{
			printf("%d\t", matriz[i*anchura + j]);
		}
		printf("\n");
	}
}

void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo)
{
	int total = ancho * alto;
	int num;

	if (modo == 'B')
	{

		for (int i = 0; i < cantidad; i++)
		{
			int r = rand() % total;
			while (matriz[r] != 0) {
				r = rand() % total;
			}

			int opcion = rand() % 100;
			if (opcion <= 50) {
				matriz[r] = 2;
			}
			else if (opcion <= 80 && opcion > 50) {
				matriz[r] = 4;
			}
			else {
				matriz[r] = 8;
			}
		}
	}
	else if (modo == 'A')
	{
		for (int i = 0; i < cantidad; i++)
		{
			int r = rand() % total;
			while (matriz[r] != 0) {
				r = rand() % total;
			}

			int opcion = rand() % 100;
			if (opcion <= 60) {
				matriz[r] = 2;
			}
			else {
				matriz[r] = 4;
			}

		}
	}



}

bool checkMove(int *matriz, int anchura, int altura)
{
	for (int i = 0; i < anchura*(altura - 1); i++)
	{
		if (matriz[i] == matriz[i + anchura] || matriz[i + anchura] == 0)
		{
			return true;
		}
	}

	for (int i = anchura; i < anchura*altura; i++)
	{
		if (matriz[i] == matriz[i - anchura] || matriz[i - anchura] == 0)
		{
			return true;
		}
	}

	for (int i = 0; i < altura; i++)
	{
		for (int j = 0; j < anchura - 1; j++)
		{
			if (matriz[i*anchura + i] == matriz[i*anchura + i + 1] || matriz[i*anchura + i + 1] == 0)
			{
				return true;
			}
		}
	}

	for (int i = 0; i < altura; i++)
	{
		for (int j = 1; j < anchura; j++)
		{
			if (matriz[i*anchura + i] == matriz[i*anchura + i - 1] || matriz[i*anchura + i - 1] == 0)
			{
				return true;
			}
		}
	}

	return false;

}

int checkFull(int *matriz, int tamano)
{
	int flag = 1;
	for (int i = 0; i < tamano; i++)
	{
		if (matriz[i] == 0)
		{
			flag = 0;
		}
	}
	return flag;
}

void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo)
{
	if (!checkFull(matriz, ancho*alto))
	{
		int n = 0;
		for (int i = 0; i < ancho*alto; i++)
		{
			if (matriz[i] == 0)
				n++;
		}
		if (modo == 'B')
		{
			if (n < 15)
			{
				generateSeeds(matriz, ancho, alto, n, modo);
			}
			else {
				generateSeeds(matriz, ancho, alto, numeroSemillas, modo);
			}

		}
		else if (modo == 'A')
		{
			if (n < 8)
			{
				generateSeeds(matriz, ancho, alto, n, modo);
			}
			else {
				generateSeeds(matriz, ancho, alto, numeroSemillas, modo);
			}

		}

	}
}

void guardar(int vidas, int *matriz, int altura, int anchura, char dificultad) {

	ofstream archivo;
	int dif;

	archivo.open("2048_savedata.txt", ios::out); //Creamos o reemplazamos el archivo

	//Si no se puede guardar ERROR
	if (archivo.fail())
	{
		cout << "Error al guardar la partida.\n";
		exit(1);
	}

	if (dificultad == 'B')
	{
		dif = 0;
	}
	else
	{
		dif = 1;
	}

	archivo << vidas << endl; //Guardamos las vidas
	archivo << altura << endl; //Guardamos las altura
	archivo << anchura << endl; //Guardamos las anchura
	archivo << dif << endl; //Guardamos la dificultad

	//Guardamos la matriz
	for (int i = 0; i < (altura*anchura); i++)
	{
		archivo << matriz[i] << " ";
	}
	cout << "\nPartida guardada con exito." << endl;

	archivo.close(); //Cerramos el archivo
}

int* cargar() {

	ifstream archivo;
	int i = 4, vidas, altura, anchura, dif;
	int *partida;

	archivo.open("2048_savedata.txt", ios::in); //Abrimos el archivo en modo lectura

	//Si no se puede cargar ERROR
	if (archivo.fail())
	{
		cout << "Error al abrir la partida guardada. El fichero no existe o est� corrupto\n";
		exit(1);
	}

	archivo >> vidas;
	archivo >> altura;
	archivo >> anchura;
	archivo >> dif;

	partida = (int*)malloc(4 * sizeof(int) + altura * anchura * sizeof(int)); //Reservamos memoria para los datos de la partida

	partida[0] = vidas; //Guardamos vidas
	partida[1] = altura; //Guardamos altura
	partida[2] = anchura; //Guardamos anchura
	partida[3] = dif; //Guardamos la dificultad

	//Guardamos la matriz
	while (!archivo.eof()) { //Mientras no sea el final del archivo
		archivo >> partida[i];
		i++;
	}

	archivo.close(); //Cerramos el archivo

	return partida;
}

int* MostrarEspecificaciones()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int * especificacion;

	especificacion = (int*)malloc(2 * sizeof(int));
	for (int i = 0; i < 2; i++) {
		especificacion[i] = 0;
	}

	especificacion[0] = prop.maxThreadsPerBlock;
	especificacion[1] = *prop.maxGridSize;

	printf("Especificaciones maximas: %d hilos/bloque %d gridsize.\n", especificacion[0], especificacion[1]);

	return especificacion;
}
